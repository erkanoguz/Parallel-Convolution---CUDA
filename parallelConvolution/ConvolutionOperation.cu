#include "hip/hip_runtime.h"
#include "ConvolutionOperation.h"


/*Get Image Operation*/
void ConvolutionOperation::imageOperaiton(ImageOperation*& ImOp)
{
	this->ImOp = ImOp;
}



/*Initializing all process*/
void ConvolutionOperation::Init(ImageOperation*& ImOp, char* path)
{
	/*In�tialize Image Operaitons*/
	imageOperaiton(ImOp);

	/*Read image from specified path*/
	ImOp->imageRead(path);

	
	/*Get Kernels*/
	kernelCol = new float[KERNEL_SIZE];
	kernelRow = new float[KERNEL_SIZE];

	ImOp->kernelImplementation();

	ImOp->getColKernel(kernelCol);
	ImOp->getRowKernel(kernelRow);
	

	/*Get image that will be convoluted*/
	image = ImOp->getImage();
	
	/*Get image size*/
	rows = image.rows;										// number of input image rows 
	cols = image.cols;										// number of input image columns
	paddedCols = rows + ceil(KERNEL_SIZE / float(2));		// number of padded image rows
	paddedRows = cols + ceil(KERNEL_SIZE / float(2));		// number of padded image columns
	
	/*Convert input image from Mat to float*/
	inputIm = new float[rows * cols];
	ImOp->Mat2Ptr(image, inputIm, rows, cols);


	/*Convert image uint to float*/
	ImOp->uint2float(image);

	/*CPU Initializing*/
	CPUInputIm = new float[paddedRows * paddedCols];
	CPUTempIm = new float[rows * cols];
	CPUOutputIm = new float[rows * cols];

	ImOp->padding(inputIm, CPUInputIm, paddedRows, paddedCols);
	ImOp->setZero(CPUTempIm, rows, cols);
	ImOp->setZero(CPUOutputIm, rows, cols);
	OutputIm = Mat::zeros(rows , cols , CV_32F);


	/*GPU Initializing*/
	kernelByteSize = sizeof(float) * KERNEL_SIZE;
	imageByteSize = sizeof(float) * rows * cols;
	paddedImageByteSize = sizeof(float) * paddedRows * paddedCols;

	GPUInputIm = new float[paddedRows * paddedCols];
	GPUTempIm = new float[rows * cols];
	GPUOutputIm = new float[rows * cols];

	ImOp->padding(inputIm, GPUInputIm, paddedRows, paddedCols);
	ImOp->setZero(GPUTempIm, rows, cols);
	ImOp->setZero(GPUOutputIm, rows, cols);

	GPUOutputImageResult = Mat::zeros(rows, cols, CV_32F);
}


void ConvolutionOperation::showResult()
{
	/*Show input image*/
	ImOp->float2uint(image);
	imshow("Input Image", image);
	
	/*Show CPU Result*/
	ImOp->float2uint(OutputIm);
	imshow("CPU Convolution Result", OutputIm);

	/*Show GPU Result*/
	ImOp->float2uint(GPUOutputImageResult);
	imshow("GPU Convolution Result", GPUOutputImageResult);
	
}



/******************************************************************************************************************************************/
/*-------------------------------------------------------CPU CONVOLUTION------------------------------------------------------------------*/
/******************************************************************************************************************************************/

void ConvolutionOperation::convolutionRowCPU(const float* source, const float* kernel, float* output, int xSize, int ySize, int kernel_size)
{

	float* temp = new float[kernel_size];
	int outRow = xSize - 2;
	int outCol = ySize - 2;

	for (int i = floor(kernel_size / 2); i < xSize - (kernel_size / 2); i++)
	{
		for (int j = floor(kernel_size / 2); j < ySize - floor(kernel_size / 2); j++)
		{
			for (int c = 0; c < 3; c++)
			{
				*(temp + c) = *(kernel + c) * *(source + i * ySize + (j + (c - kernel_size + 2)));
			}
			*(output + (i - 1) * outCol + (j - 1)) = *(temp + 0) + *(temp + 1) + *(temp + 2);
		}
	}

}

void ConvolutionOperation::convolutionColCPU(const float* source, const float* kernel, float* output, int xSize, int ySize, int kernel_size)
{

	float* temp = new float[kernel_size];
	int outRow = xSize - 2;
	int outCol = ySize - 2;

	for (int i = floor(kernel_size / 2); i < xSize - (kernel_size / 2); i++)
	{
		for (int j = floor(kernel_size / 2); j < ySize - floor(kernel_size / 2); j++)
		{
			for (int c = 0; c < 3; c++)
			{
				*(temp + c) = *(kernel + c) * *(source + (i + (c - kernel_size + 2)) * ySize + j);
			}
			*(output + (i - 1) * outCol + (j - 1)) = *(temp + 0) + *(temp + 1) + *(temp + 2);
		}
	}

}

void ConvolutionOperation::runCPUConvolution()
{

	auto startTime = high_resolution_clock::now();
	convolutionColCPU(CPUInputIm, kernelCol, CPUTempIm, paddedRows, paddedCols, KERNEL_SIZE);
	ImOp->padding(CPUTempIm, CPUInputIm, paddedRows, paddedCols);
	convolutionRowCPU(CPUInputIm, kernelRow, CPUOutputIm, paddedRows, paddedCols, KERNEL_SIZE);
	auto endTime = high_resolution_clock::now();
	CPUElapsedTime = endTime - startTime;
	cout << "CPU elapsed time: " << CPUElapsedTime.count() << endl;
	
	/*Float to Mat output image*/
	ImOp->Ptr2Mat(CPUOutputIm, OutputIm, rows, cols);
	

}


/******************************************************************************************************************************************/
/*-------------------------------------------------------GPU CONVOLUTION------------------------------------------------------------------*/
/******************************************************************************************************************************************/

__global__ void GPUconvolutionCol(float* source, float* kernel, float* output, int xSize, int ySize, int kernel_size)
{

	int outRow = xSize - 2;
	int outCol = ySize - 2;

	int i = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;

	if (i >= 1 && i < xSize - 1 && j >= 1 && j < ySize - 1)
	{
		*(output + (i - 1) * outCol + (j - 1)) = 0;
		for (int c = 0; c < 3; c++)
		{
			*(output + (i - 1) * outCol + (j - 1)) += *(kernel + c) * *(source + (i + (c - kernel_size + 2)) * ySize + j);
		}
	}
}

__global__ void paddingImageGPU(float* source, float* padded, int xPaddedSize, int yPaddedSize)
{

	int ySize = yPaddedSize - 2;
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockIdx.y * blockDim.y + 1;


	if (i >= 1 && i < xPaddedSize - 1 && j >= 1 && j < xPaddedSize - 1)
	{
		*(padded + i * yPaddedSize + j) = *(source + (i - 1) * ySize + (j - 1));
	}
}

__global__ void GPUconvolutionRow(float* source, float* kernel, float* output, int xSize, int ySize, int kernel_size)
{

	//float* temp = new float[kernel_size];
	int outRow = xSize - 2;
	int outCol = ySize - 2;

	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockIdx.y * blockDim.y + 1;

	if (i >= 1 && i < xSize - 1 && j >= 1 && j < ySize - 1)
	{
		*(output + (i - 1) * outCol + (j - 1)) = 0;
		for (int c = 0; c < 3; c++)
		{
			*(output + (i - 1) * outCol + (j - 1)) += *(kernel + c) * *(source + i * ySize + (j + (c - kernel_size + 2)));
		}
	}
}

void ConvolutionOperation::runGPUConvolution()
{

	/*Allocate device memory*/
	hipMalloc((void**)&dev_inputIm, paddedImageByteSize);
	hipMalloc((void**)&dev_tempIm, imageByteSize);
	hipMalloc((void**)& dev_outputIm, imageByteSize);
	hipMalloc((void**)&dev_kernelRow, kernelByteSize);
	hipMalloc((void**)& dev_kernelCol, kernelByteSize);
	
	/*Transfer Variable from gost to device*/
	hipMemcpy(dev_inputIm, GPUInputIm, paddedImageByteSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_tempIm, GPUTempIm, imageByteSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_outputIm, GPUOutputIm, imageByteSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernelRow, kernelRow, kernelByteSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernelCol, kernelCol, kernelByteSize, hipMemcpyHostToDevice);
	
	/*CUDA configuration*/

	int xDim = 16;
	int yDim = 16;
	dim3 block(xDim, yDim);
	dim3 grid((paddedRows + xDim - 1) / xDim, (paddedCols + yDim - 1) / yDim);

	auto startTime = high_resolution_clock::now();
	GPUconvolutionCol << <grid, block >> > (dev_inputIm, dev_kernelCol, dev_tempIm, paddedRows, paddedCols, KERNEL_SIZE);
	paddingImageGPU << <grid, block >> > (dev_tempIm, dev_inputIm, paddedRows, paddedCols);
	GPUconvolutionRow << <grid, block >> > (dev_inputIm, dev_kernelRow, dev_outputIm, paddedRows, paddedCols, KERNEL_SIZE);
	auto endTime = high_resolution_clock::now();

	duration <double> GPUElapsedTime = endTime - startTime;

	cout << "GPU elapsed time: " << GPUElapsedTime.count() << endl;

	/*GPU convolution result*/
	hipMemcpy(GPUOutputIm, dev_outputIm, imageByteSize, hipMemcpyDeviceToHost);

	ImOp->Ptr2Mat(GPUOutputIm, GPUOutputImageResult, rows, cols);

}

